#include "hip/hip_runtime.h"
#include "pbd.cuh"

__global__ void pbd_kernel(Particle* particles, int numParticles, float dt, int numIterations) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        // Simple gravity
        particles[i].vel.y -= 9.81f * dt;

        // Simple collision with a plane at y = 0
        if (particles[i].pos.y < 0.0f) {
            particles[i].pos.y = 0.0f;
        }


        // Basic iterative position-based solver (very simplified)
        for(int j = 0; j < numIterations; ++j){
            particles[i].pos.y = fmaxf(particles[i].pos.y, 0.0f); // fmaxfを使う
        }

        //Update velocity (very basic, no proper integration)
        particles[i].vel.x = (particles[i].pos.x - particles[i].oldPos.x) / dt;
        particles[i].vel.y = (particles[i].pos.y - particles[i].oldPos.y) / dt;
        particles[i].vel.z = (particles[i].pos.z - particles[i].oldPos.z) / dt;
        particles[i].oldPos = particles[i].pos;
    }
}

void pbd_step(Particle* particles, int numParticles, float dt, int numIterations) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;
  pbd_kernel<<<blocksPerGrid, threadsPerBlock>>>(particles, numParticles, dt, numIterations);
  hipDeviceSynchronize();
}